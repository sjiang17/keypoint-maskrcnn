#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * \file roi_pooling.cu
 * \brief roi pooling operator
 * \author Ross Girshick, Kye-Hyeon Kim, Jian Guo
*/
#include "./roi_pooling_v1-inl.h"
#include "../common/cuda_utils.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <algorithm>
#include <vector>

#define CUDA_1D_KERNEL_LOOP(i, n)                                 \
for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
     i += blockDim.x * gridDim.x)

constexpr int CAFFE_CUDA_NUM_THREADS = 512;
constexpr int CAFFE_MAXIMUM_NUM_BLOCKS = 4096;

inline int CAFFE_GET_BLOCKS(const int N) {
  return std::min((N + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS,
                  CAFFE_MAXIMUM_NUM_BLOCKS);
}

namespace mshadow {
namespace cuda {

template <typename T>
__global__ void ROIPoolForward(
    const int nthreads,
    const T* bottom_data,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const T* bottom_rois,
    T* top_data,
    T* argmax_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    int roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
    int roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
    int roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
    int roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    int hstart = static_cast<int>(floor(static_cast<T>(ph) * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw) * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1) * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1) * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    T maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxidx = -1;
    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h * width + w;
        if (offset_bottom_data[bottom_index] > maxval) {
          maxval = offset_bottom_data[bottom_index];
          maxidx = bottom_index;
        }
      }
    }
    top_data[index] = maxval;
    if (argmax_data) {
      argmax_data[index] = maxidx;
    }
  }
}

template <typename T>
__global__ void ROIPoolBackward(
    const int nthreads,
    const T* top_diff,
    const T* argmax_data,
    const int num_rois,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    int bottom_offset = (roi_batch_ind * channels + c) * height * width;
    int top_offset = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    T* offset_bottom_diff = bottom_diff + bottom_offset;
    const T* offset_argmax_data = argmax_data + top_offset;

    int argmax = static_cast<int>(offset_argmax_data[ph * pooled_width + pw]);
    if (argmax != -1) {
      atomicAdd(offset_bottom_diff + argmax, static_cast<T>(offset_top_diff[ph * pooled_width + pw]));
    }
  }
}

template<typename Dtype>
inline void ROIPoolForward_v1(const Tensor<gpu, 4, Dtype> &out,
                           const Tensor<gpu, 4, Dtype> &data,
                           const Tensor<gpu, 2, Dtype> &bbox,
                           const Tensor<gpu, 4, Dtype> &max_idx,
                           const float spatial_scale) {
  const Dtype *bottom_data = data.dptr_;
  const Dtype *bottom_rois = bbox.dptr_;
  Dtype *top_data = out.dptr_;
  Dtype *argmax_data = max_idx.dptr_;
  const int count = out.shape_.Size();
  const int channels = data.size(1);
  const int height = data.size(2);
  const int width = data.size(3);
  const int pooled_height = out.size(2);
  const int pooled_width = out.size(3);
  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridDim, (gridSize + kMaxGridDim - 1) / kMaxGridDim);
  dim3 dimBlock(kMaxThreadsPerBlock);
  CheckLaunchParam(dimGrid, dimBlock, "ROIPooling Forward");
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);

  ROIPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
      count, bottom_data, spatial_scale, channels, height, width,
      pooled_height, pooled_width, bottom_rois, top_data, argmax_data);
}

template<typename Dtype>
inline void ROIPoolBackwardAcc_v1(const Tensor<gpu, 4, Dtype> &in_grad,
                               const Tensor<gpu, 4, Dtype> &out_grad,
                               const Tensor<gpu, 2, Dtype> &bbox,
                               const Tensor<gpu, 4, Dtype> &max_idx,
                               const float spatial_scale) {
  const Dtype *top_diff = out_grad.dptr_;
  const Dtype *bottom_rois = bbox.dptr_;
  Dtype *bottom_diff = in_grad.dptr_;
  Dtype *argmax_data = max_idx.dptr_;
  const int count = in_grad.shape_.Size();
  const int num_rois = bbox.size(0);
  const int channels = in_grad.size(1);
  const int height = in_grad.size(2);
  const int width = in_grad.size(3);
  const int pooled_height = out_grad.size(2);
  const int pooled_width = out_grad.size(3);
  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridDim, (gridSize + kMaxGridDim - 1) / kMaxGridDim);
  dim3 dimBlock(kMaxThreadsPerBlock);
  CheckLaunchParam(dimGrid, dimBlock, "ROIPooling Backward");
  hipStream_t stream = Stream<gpu>::GetStream(in_grad.stream_);

  ROIPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(out_grad.shape_.Size()), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
    out_grad.shape_.Size(), top_diff, argmax_data, num_rois, spatial_scale, channels, height, width,
      pooled_height, pooled_width, bottom_diff, bottom_rois);
}

}  // namespace cuda

template<typename Dtype>
inline void ROIPoolForward_v1(const Tensor<gpu, 4, Dtype> &out,
                           const Tensor<gpu, 4, Dtype> &data,
                           const Tensor<gpu, 2, Dtype> &bbox,
                           const Tensor<gpu, 4, Dtype> &max_idx,
                           const float spatial_scale) {
  cuda::ROIPoolForward_v1(out, data, bbox, max_idx, spatial_scale);
}

template<typename Dtype>
inline void ROIPoolBackwardAcc_v1(const Tensor<gpu, 4, Dtype> &in_grad,
                               const Tensor<gpu, 4, Dtype> &out_grad,
                               const Tensor<gpu, 2, Dtype> &bbox,
                               const Tensor<gpu, 4, Dtype> &max_idx,
                               const float spatial_scale) {
  cuda::ROIPoolBackwardAcc_v1(in_grad, out_grad, bbox, max_idx, spatial_scale);
}

}  // namespace mshadow


namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(ROIPoolingParam_v1 param, int dtype) {
  Operator* op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new ROIPoolingOp_v1<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
